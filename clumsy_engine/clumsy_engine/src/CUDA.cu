#include "hip/hip_runtime.h"


#include "CUDA.h"
#include "hip/hip_runtime.h" 
#include <iostream>

#include "matrix_math/matrix_math.h"

#include "raw_pointer_kernels.h"


namespace clumsy_engine
{

	template< typename Kernel > __global__ void cuda_for_loop(Kernel k, int size)
	{
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid < size)
		{
			//printf(" in cuda %d \n", tid);
			k(tid);
		}
	}

	template<typename Kernel>
	void CUDA::loop(Kernel k, int size)
	{
		int tpb = 128;
		int bpg = (size - 1) / tpb + 1;
		cuda_for_loop << <bpg, tpb >> > (k, size);
		//cuda_for_loop << <1, 1 >> > (k, size);
	}

	template void CUDA::loop<Copy<float, float>>(Copy<float, float> k, int size);
	template void CUDA::loop<Copy<vec3f, vec3f>>(Copy<vec3f, vec3f> k, int size);

	template void CUDA::loop<DummyMove>(DummyMove k, int size);
}


