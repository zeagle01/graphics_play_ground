#include "hip/hip_runtime.h"


#include "CUDA.h"
#include "hip/hip_runtime.h" 
#include <iostream>

#include "raw_pointer_kernels.h"


namespace clumsy_engine
{

	template< typename Kernel > __global__ void cuda_for_loop(Kernel k, int size)
	{
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid < size)
		{
			k(tid);
		}
	}

	template<typename Kernel>
	void CUDA::loop(Kernel k, int size)
	{
		int tpb = 128;
		int bpg = (size - 1) / tpb + 1;
		cuda_for_loop << <bpg, tpb >> > (k, size);
		//cuda_for_loop << <1, 1 >> > (k, size);
	}

	template void CUDA::loop<Copy<float, float>>(Copy<float, float> k, int size);
}


